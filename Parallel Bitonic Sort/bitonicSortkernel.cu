#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime_api.h>

#include "writeToCSVFileHeader.h"
#include "userInputHeader.h"

void printArray(int *elements);


int deviceBlocks;
int threadsPerBlock;
int elementsToSort;
int threadCount;


const int randMax = 10000;

void createUnsortedArray(int* elements){

	//Get size and cuda dimentions from user input

	for (int i = 0; i < elementsToSort; ++i){
		elements[i] = rand() % randMax - rand() % 5;
	}

}

bool isSorted(int *elements){

	bool sorted = true;
	for (int i = 0; i < (elementsToSort - 1); ++i){
		if (elements[i] > elements[i + 1]){
			sorted = false;
		}
	}
	return sorted;
}


void print_elapsed(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.3fs\n", elapsed);
}

int random_int()
{
	return (int)rand() / (int)2048;
}

/*
We get our indexes to swap buy xoring our thread index with the step. This essentially wraps the thread Id round the step value meaning 
the only values porduced that are bigger than the Thread Id will be those within the requied step length
*/
__global__ void stepskernel(int *dev_values, int step, int phaseLength)
{
	unsigned int firstIndex, XoredSecondIndex;
	//Set it to the thread Id
	firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	XoredSecondIndex = firstIndex ^ step;

	//Threads i corrasponding to the desired bitonic element will be used for the swap
	if ((XoredSecondIndex)>firstIndex) {

		if ((firstIndex&phaseLength) == 0) {
			if (dev_values[firstIndex]>dev_values[XoredSecondIndex]) {
		
				int temp = dev_values[firstIndex];
				dev_values[firstIndex] = dev_values[XoredSecondIndex];
				dev_values[XoredSecondIndex] = temp;
			}
		}
		if ((firstIndex&phaseLength) != 0) {

			if (dev_values[firstIndex]<dev_values[XoredSecondIndex]) {
				int temp = dev_values[firstIndex];
				dev_values[firstIndex] = dev_values[XoredSecondIndex];
				dev_values[XoredSecondIndex] = temp;
			}
		}
	}
}

/*
Main function call. Created array and calls stepskernel based of the size of the bitonic sequences and step.
*/
void bitonic_sort(int *values)
{
	int *dev_values;
	size_t size = elementsToSort* sizeof(int);

	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	dim3 blocks(deviceBlocks, 1);   
	dim3 threads(threadCount, 1);  

	int step, phaseLength;

	for (phaseLength = 2; phaseLength <= elementsToSort; phaseLength <<= 1) {

		for (step = phaseLength >> 1; step>0; step = step >> 1) {
			stepskernel << <blocks, threads >> >(dev_values, step, phaseLength);
		}
	}

	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}

int main(void)
{
	int* Allthreads;
	int* AllBlocks;
	int* allTimes;
	char* allResults;

	bool runSort = true;

	while (runSort){

		clock_t start, stop;

		//Get thread, blocks and  element count
		elementsToSort = getElementCount();
		deviceBlocks = getBlockCount();
		threadsPerBlock = getThreadCount();

		threadCount = threadsPerBlock * deviceBlocks;

		//Malloc array, add values to it and write unsorted array to csv file
		int* values	 = (int*)malloc(elementsToSort*sizeof(int));
		createUnsortedArray(values);
		writeBlockElementCsvFile(values, "preSorted", threadCount, deviceBlocks);

		//Do Sort and time it
		start = clock();
		bitonic_sort(values);
		stop = clock();

		print_elapsed(start, stop);

		char* arrayState;

		if (isSorted(values)){

			printf("Is Sorted \n");
			arrayState = "sorted";
		}
		else{

			printf("Not Sorted \n");
			arrayState = "unsorted";
		}

		writeBlockElementCsvFile(values, arrayState, threadCount, deviceBlocks);

		free(values);

		runSort = runSortAgain();
	}


	getchar();
}